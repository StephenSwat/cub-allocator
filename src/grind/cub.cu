#include <hipcub/hipcub.hpp>

int main() {
    hipcub::CachingDeviceAllocator allocator;

    void *d;

    for (int i = 0; i < 100000; i++) {
        allocator.DeviceAllocate(&d, 1024);
        allocator.DeviceFree(d);
    }
}
