#include <hipcub/hipcub.hpp>

int main() {
    hipcub::CachingDeviceAllocator allocator(8, 3, 7, 1073741824, false, true);

    void *d;

    allocator.DeviceAllocate(&d, 512);
    allocator.DeviceAllocate(&d, 1024);
    allocator.DeviceAllocate(&d, 1024);
    allocator.DeviceAllocate(&d, 4096);
    allocator.DeviceAllocate(&d, 262144);
    allocator.DeviceAllocate(&d, 524288);
}
