#include <hipcub/hipcub.hpp>

int main() {
    hipcub::CachingDeviceAllocator allocator(8, 4, 4, 1073741824, false, true);

    void *d;

    allocator.DeviceAllocate(&d, 8192);
    allocator.DeviceFree(d);

    allocator.DeviceAllocate(&d, 8192);
    allocator.DeviceFree(d);
}
