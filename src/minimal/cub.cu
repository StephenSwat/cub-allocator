#include <hipcub/hipcub.hpp>

int main() {
    hipcub::CachingDeviceAllocator allocator;

    void *d;

    allocator.DeviceAllocate(&d, 1024);
    allocator.DeviceFree(d);
}
